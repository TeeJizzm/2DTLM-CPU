// Includes
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <ctime>

// Definitions
#define M_PI 3.14276
#define c 299792458
#define mu0 M_PI*4e-7
#define eta0 c*mu0



// CPU function for source calculation
void stageSource(double* V1, double* V2, double* V3, double* V4, int x, int y, double E0, int NY) {
    /* Stage 1: Source */

    // Adapted to be 1D
    V1[x * NY + y] = V1[x * NY + y] + E0;
    V2[x * NY + y] = V2[x * NY + y] - E0;
    V3[x * NY + y] = V3[x * NY + y] - E0;
    V4[x * NY + y] = V4[x * NY + y] + E0;
    // Using 1 dimensional arrays is more obvious to work with when porting to GPU

} // end func

// CPU function
void stageScatter(double* V1, double* V2, double* V3, double* V4, int NX, int NY, double Z) {
    /* Stage 2: Scatter */
    // Variables 
    double I = 0, V = 0;

    // Parallelisable code

    // for int i = 0; i < NX*NY; i++
    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            I = (V1[(x * NY) + y] + V4[(x * NY) + y] - V2[(x * NY) + y] - V3[(x * NY) + y]) / (2);
            // factorized by 2 for unnecessary mathematics

            V = 2 * V1[x * NY + y] - I;         //port1
            V1[x * NY + y] = V - V1[x * NY + y];

            V = 2 * V2[x * NY + y] + I;         //port2
            V2[x * NY + y] = V - V2[x * NY + y];

            V = 2 * V3[x * NY + y] + I;         //port3
            V3[x * NY + y] = V - V3[x * NY + y];

            V = 2 * V4[x * NY + y] - I;         //port4
            V4[x * NY + y] = V - V4[x * NY + y];
        }
    }
} // end func



// CPU Function
void stageConnect(double* V1, double* V2, double* V3, double* V4, // Arrays
    int NX, int NY, // Array arguments
    double rXmin, double rXmax, double rYmin, double rYmax) { // Boundary conditions
/* Stage 3: Connect */
// Variables
    double tempV = 0;

    // Connect internals
    for (int x = 1; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            tempV = V2[x * NY + y];
            V2[x * NY + y] = V4[(x - 1) * NY + y];
            V4[(x - 1) * NY + y] = tempV;
        }
    }
    for (int x = 0; x < NX; x++) {
        for (int y = 1; y < NY; y++) {
            tempV = V1[x * NY + y];
            V1[x * NY + y] = V3[x * NY + y - 1];
            V3[x * NY + y - 1] = tempV;
        }
    }

    // Connect boundaries
    for (int x = 0; x < NX; x++) {
        V3[x * NY + NY - 1] = rYmax * V3[x * NY + NY - 1];
        V1[x * NY] = rYmin * V1[x * NY]; // V1[x * NY + 0] = rYmin * V1[x * NY + 0];
    }
    for (int y = 0; y < NY; y++) {
        V4[(NX - 1) * NY + y] = rXmax * V4[(NX - 1) * NY + y];
        V2[y] = rXmin * V2[y]; // V2[0 * NY + y] = rXmin * V2[0 * NY + y];
    }
} // end func



int main() {

    // Start timer
    std::clock_t start = std::clock();

    /* Variables */
    // Changable variables
    int NX = 200; // number of X
    int NY = 200; // number of Y
    int NT = 8192; // number of Times/Iterations
    double dl = 1;

    // Retrieval from GPU
    //*/
    double* V1 = new double[int(NX * NY)](); // new double[int(NX*NY)](); // Sets all values to 0 
    double* V2 = new double[int(NX * NY)]();
    double* V3 = new double[int(NX * NY)]();
    double* V4 = new double[int(NX * NY)]();
    //*/


    // Variables and coefficients
    // Scatter coefficient
    double Z = eta0 / sqrt(2.);

    // Boundary connect Coefficiants
    double rXmin = -1;
    double rXmax = -1;
    double rYmin = -1;
    double rYmax = -1;

    // impulse parameters
    double dt = dl / (sqrt(2.) * c);
    double width = 20 * dt * sqrt(2.);
    double delay = 100 * dt * sqrt(2.);

    // input position
    int Ein[] = { 10,10 };
    // output/reading position
    int Eout[] = { 15,15 };

    // file output
    std::ofstream output("CPU.csv");


    for (int n = 0; n < NT; n++) {
        // Variables dependant on n
        double E0 = (1 / sqrt(2.)) * exp(-(n * dt - delay) * (n * dt - delay) / (width * width));


        /* Stage 1: Source */
        stageSource(V1, V2, V3, V4, Ein[0], Ein[1], E0, NY);


        /* Stage 2: Scatter */
        stageScatter(V1, V2, V3, V4, NX, NY, Z);
        

        /* Stage 3: Connect */
        stageConnect(V1, V2, V3, V4, NX, NY, rXmin, rXmax, rYmin, rYmax);
      

        output << n * dt << "," << V2[Eout[0] * NY + Eout[1]] + V4[Eout[0] * NY + Eout[1]] << std::endl;
        if (n % 100 == 0)
            std::cout << n << std::endl;

    } // End of loop


    output.close();
    std::cout << "Done: " << ((std::clock() - start) / (double)CLOCKS_PER_SEC) << std::endl;
    std::cin.get();


} // end main

// EOF